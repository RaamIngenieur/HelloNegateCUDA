#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <png.hpp>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


class UCarray
{
public:
	unsigned char *Pix;
	UCarray(int r, int c);
	~UCarray();
	void ImagetoArray(png::image<png::gray_pixel>* input);
	void operator=(const UCarray& other);
	void ArraytoImage(png::image<png::gray_pixel>* output);
private:
	int row, column;

};

__global__ void negateKernel(unsigned char* img, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<N)
	img[i] = 255 - img[i];
}

UCarray::UCarray(int r, int c)
{
	row = r;
	column = c;
	Pix = (unsigned char*)malloc(row * column * sizeof(unsigned char));
	if (Pix == NULL)
	{
		fprintf(stderr, "out of memory\n");
	}

}

UCarray::~UCarray()
{
	free(Pix);
}

void UCarray::ImagetoArray(png::image<png::gray_pixel>* input)
{
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
		{
			*(Pix + i*column + j) = (*input)[i][j];
		}
	}
}

void UCarray::ArraytoImage(png::image<png::gray_pixel>* output)
{
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
		{
			(*output)[i][j] = *(Pix + i*column + j);
		}
	}
}

void UCarray:: operator =(const UCarray&  other)
{
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < column; j++)
		{
			*(Pix + i*column + j) = *(other.Pix + i*column + j);
		}
	}
}


int main()
{

	png::image< png::gray_pixel > image("tsukuba_daylight_L_00001.png");

	int row = image.get_height(), column = image.get_width(), N= row*column;

	UCarray UCInput(row, column);

	UCInput.ImagetoArray(&image);

	unsigned char *d_x;

	hipMalloc(&d_x, N*sizeof(unsigned char));

	hipMemcpy(d_x, UCInput.Pix, N*sizeof(unsigned char), hipMemcpyHostToDevice);

	negateKernel <<<(N + 255) / 256, 256 >>>(d_x, N);

	hipMemcpy(UCInput.Pix, d_x, N*sizeof(unsigned char), hipMemcpyDeviceToHost);

	UCInput.ArraytoImage(&image);

	image.write("negated.png");

    return 0;
}

